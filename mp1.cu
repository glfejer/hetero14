#include "hip/hip_runtime.h"
// MP 1
#include "hip/hip_runtime.h"
#include ""

#include	<wb.h>
#include "wbCheck.h"

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < len)
    {
        out[i] = in1[i] + in2[i];
    }
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;
    float * hostIncpy1;
    float * hostIncpy2;

    int devCount = -1;
    int curDevice = -2;
    hipGetDeviceCount(&devCount);
    hipGetDevice(&curDevice);
    wbLog(TRACE, "device count=", devCount, "\t\tcurDevice=", curDevice);
    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *)malloc(inputLength * sizeof(float));
    hostIncpy1 = (float *)malloc(inputLength * sizeof(float));
    hostIncpy2 = (float *)malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    int size = inputLength * sizeof(float);
    wbLog(TRACE, "inLength=", inputLength, " size=", size);
    wbCheck (hipMalloc((void**)&deviceInput1, size));
    wbCheck(hipMalloc((void**)&deviceInput2, size));
    wbCheck(hipMalloc((void**)&deviceOutput, size));


    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);

    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    int blockx = 256;
    //int numblks = ceil(inputLength / (float)blockx);
    int numblks = (inputLength + blockx - 1) / blockx;
    wbLog(TRACE, "blockx=", blockx, " numblks=", numblks);

    dim3 DimGrid(numblks, 1, 1);
    dim3 DimBlock(blockx, 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    //@@ Launch the GPU Kernel here
    //vecAdd <<<numblks, blockx>>> (deviceInput1, deviceInput2, deviceOutput, inputLength);
    vecAdd <<<DimGrid, DimBlock >>> (deviceInput1, deviceInput2, deviceOutput, inputLength);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
    hipMemcpy(hostIncpy1, deviceInput1, size, hipMemcpyDeviceToHost);
    hipMemcpy(hostIncpy2, deviceInput2, size, hipMemcpyDeviceToHost);

    for (int ix = 0; ix < inputLength; ix++)
    {
        cout << hostIncpy1[ix] << "\t" << hostIncpy2[ix] << "\t" << hostOutput[ix] << endl;
    }

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

